#include "gpu.h"

#include "MeshView.h"

#include <iostream>
#include <cstdlib>
#include <algorithm>

#define CUDA_CHECK(x) do { hipError_t __err = (x);\
    if (__err != hipSuccess) { std::cerr << __FILE__ << ":" << __LINE__ << " CUDA call `" << #x \
        << "' failed with error `" << hipGetErrorString(__err) << std::endl; abort(); } \
} while (false)

GPUMeshView::GPUMeshView(MeshView &mv) {
    nP = mv.pts.size();
    int nT = mv.tets.size();

    CUDA_CHECK(hipMalloc(&pts,    nP * sizeof(point)));
    CUDA_CHECK(hipMalloc(&anyTet, nP * sizeof(int)  ));
    CUDA_CHECK(hipMalloc(&tets,   nT * sizeof(tet)  ));
    CUDA_CHECK(hipMalloc(&kappa,  nT * sizeof(real) ));
    CUDA_CHECK(hipMalloc(&Ip,     nT * sizeof(real) ));
}

GPUMeshView::~GPUMeshView() {
    CUDA_CHECK(hipFree(pts));
    CUDA_CHECK(hipFree(anyTet));
    CUDA_CHECK(hipFree(tets));
    CUDA_CHECK(hipFree(kappa));
    CUDA_CHECK(hipFree(Ip));
}

GPUAverageSolution::GPUAverageSolution(const MeshView &mv) : U(nP) {
    nP = mv.pts.size();
    CUDA_CHECK(hipMalloc(&Udev, nP * sizeof(real)));
}

GPUAverageSolution::~GPUAverageSolution() {
    CUDA_CHECK(hipFree(Udev));
}

std::vector<double> &GPUAverageSolution::retrieve() {
    std::vector<real> Uhost(nP);
    CUDA_CHECK(hipMemcpy(Uhost.data(), Udev, nP * sizeof(real), hipMemcpyDeviceToHost));
    std::copy(Uhost.begin(), Uhost.end(), U.begin());
    return U;
}
