#include "gpu.h"
#include "kernels.h"

#include "MeshView.h"

#include <iostream>
#include <cstdlib>
#include <algorithm>
#include <cassert>

#include <hipblas.h>

#define CUDA_CHECK(x) do { hipError_t __err = (x);\
    if (__err != hipSuccess) { std::cerr << __FILE__ << ":" << __LINE__ << " CUDA call `" << #x \
        << "' failed with error `" << hipGetErrorString(__err) << "'" << std::endl; abort(); } \
} while (false)

void CUBLASWINAPI cublasXaxpy(int n, float alpha, const float *x, int incx, float *y, int incy) {
    return hipblasSaxpy(n, alpha, x, incx, y, incy);
}

void CUBLASWINAPI cublasXaxpy(int n, double alpha, const double *x, int incx, double *y, int incy) {
    return hipblasDaxpy(n, alpha, x, incx, y, incy);
}

GPUMeshView::GPUMeshView(int rank, int device, MeshView &mv) {
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, device));
    CUDA_CHECK(hipSetDevice(device));
    std::cout << "Rank " << rank << " using device #" << device << " (" << prop.name << ")" << std::endl;

    nP = mv.pts.size();
    int nT = mv.elems.size();

    CUDA_CHECK(hipMalloc(&pts,    nP * sizeof(point)));
    CUDA_CHECK(hipMalloc(&anyTet, nP * sizeof(int)));
    CUDA_CHECK(hipMalloc(&elems,  nT * sizeof(MeshElement)));

    CUDA_CHECK(hipMemcpy(pts,    mv.pts   .data(), nP * sizeof(point),       hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(anyTet, mv.anyTet.data(), nP * sizeof(int),         hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(elems,  mv.elems .data(), nT * sizeof(MeshElement), hipMemcpyHostToDevice));
}

GPUMeshView::~GPUMeshView() {
    CUDA_CHECK(hipFree(pts));
    CUDA_CHECK(hipFree(anyTet));
    CUDA_CHECK(hipFree(elems));
}

GPUAverageSolution::GPUAverageSolution(const GPUMeshView &gmv) : nP(gmv.nP), U(nP * NFREQ) {
    hipblasStatus_t status = cublasInit();
    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "CUBLAS init failed with status " << status << std::endl;
        abort();
    }
    CUDA_CHECK(hipMalloc(&Udev, nP * NFREQ * sizeof(real)));
    CUDA_CHECK(hipMemset(Udev, 0, nP * NFREQ * sizeof(real)));
}

GPUAverageSolution::~GPUAverageSolution() {
    CUDA_CHECK(hipFree(Udev));
    cublasShutdown();
}

const std::vector<real> &GPUAverageSolution::retrieve() {
    CUDA_CHECK(hipMemcpy(U.data(), Udev, nP * NFREQ * sizeof(real), hipMemcpyDeviceToHost));
    return U;
}

void GPUAverageSolution::add(real *Idir, const real wei) {
    cublasXaxpy(nP * NFREQ, wei, Idir, 1, Udev, 1);
}

GPUMultipleDirectionSolver::GPUMultipleDirectionSolver(
        const int maxDirections, const GPUMeshView &mv, const std::vector<point> &ws
)
    : maxDirections(maxDirections), mv(mv)
{
    CUDA_CHECK(hipMalloc(&Idirs, mv.nP * NFREQ * maxDirections * sizeof(real)));
    CUDA_CHECK(hipMalloc(&inner, mv.nP * maxDirections * sizeof(int)));
    CUDA_CHECK(hipMalloc(&w, ws.size() * sizeof(point)));
    CUDA_CHECK(hipMemcpy(w, ws.data(), ws.size() * sizeof(point), hipMemcpyHostToDevice));
}

real *GPUMultipleDirectionSolver::Idir(const int direction) {
    assert(direction < maxDirections);
    return Idirs + direction * mv.nP * NFREQ;
}

int *GPUMultipleDirectionSolver::innerFlag(const int direction) {
    assert(direction < maxDirections);
    return inner + direction * mv.nP;
}

GPUMultipleDirectionSolver::~GPUMultipleDirectionSolver() {
    CUDA_CHECK(hipFree(Idirs));
    CUDA_CHECK(hipFree(inner));
    CUDA_CHECK(hipFree(w));
}

void GPUMultipleDirectionSolver::setBoundary(
        const int direction, std::vector<real> &Ihostdir, std::vector<int> &isInner)
{
    CUDA_CHECK(hipMemcpy(Idir(direction), Ihostdir.data(), mv.nP * NFREQ * sizeof(real), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(innerFlag(direction), isInner.data(), mv.nP * sizeof(int), hipMemcpyHostToDevice));
}

void GPUMultipleDirectionSolver::traceInterior(const int lo, const int offs, const int ndir) {
    const int nP = mv.nP;

    dim3 block(NFREQ, PTSPERBLOCK);
    dim3 grid((nP + PTSPERBLOCK - 1) / PTSPERBLOCK, ndir);

    trace_kernel<<<grid, block>>>(nP, lo, offs, mv, Idirs, inner, w);
}
