#include "gpu.h"

#include "MeshView.h"

#include <iostream>
#include <cstdlib>
#include <algorithm>
#include <cassert>

#include <hipblas.h>

#define CUDA_CHECK(x) do { hipError_t __err = (x);\
    if (__err != hipSuccess) { std::cerr << __FILE__ << ":" << __LINE__ << " CUDA call `" << #x \
        << "' failed with error `" << hipGetErrorString(__err) << std::endl; abort(); } \
} while (false)

GPUMeshView::GPUMeshView(MeshView &mv) {
    nP = mv.pts.size();
    int nT = mv.tets.size();

    CUDA_CHECK(hipMalloc(&pts,    nP * sizeof(point)));
    CUDA_CHECK(hipMalloc(&anyTet, nP * sizeof(int)  ));
    CUDA_CHECK(hipMalloc(&tets,   nT * sizeof(tet)  ));
    CUDA_CHECK(hipMalloc(&kappa,  nT * sizeof(real) ));
    CUDA_CHECK(hipMalloc(&Ip,     nT * sizeof(real) ));
}

GPUMeshView::~GPUMeshView() {
    CUDA_CHECK(hipFree(pts));
    CUDA_CHECK(hipFree(anyTet));
    CUDA_CHECK(hipFree(tets));
    CUDA_CHECK(hipFree(kappa));
    CUDA_CHECK(hipFree(Ip));
}

GPUAverageSolution::GPUAverageSolution(const MeshView &mv) : nP(mv.pts.size()), U(nP) {
    cublasInit();
    CUDA_CHECK(hipMalloc(&Udev, nP * sizeof(real)));
    CUDA_CHECK(hipMemset(Udev, 0, nP * sizeof(real)));
}

GPUAverageSolution::~GPUAverageSolution() {
    CUDA_CHECK(hipFree(Udev));
    cublasShutdown();
}

std::vector<double> &GPUAverageSolution::retrieve() {
    std::vector<real> Uhost(nP);
    CUDA_CHECK(hipMemcpy(Uhost.data(), Udev, nP * sizeof(real), hipMemcpyDeviceToHost));
    std::copy(Uhost.begin(), Uhost.end(), U.begin());
    return U;
}

template<>
void GPUAverageSolution::add<float>(float *Idir, const float wei) {
    assert(sizeof(real) == sizeof(float));
    hipblasSaxpy(nP, wei, Idir, 1, (float *)Udev, 1);
}

template<>
void GPUAverageSolution::add<double>(double *Idir, const double wei) {
    assert(sizeof(real) == sizeof(double));
    hipblasDaxpy(nP, wei, Idir, 1, (double *)Udev, 1);
}

GPUMultipleDirectionSolver::GPUMultipleDirectionSolver(const int maxDirections, const GPUMeshView &mv)
    : maxDirections(maxDirections), mv(mv)
{
    CUDA_CHECK(hipMalloc(&Idirs, mv.nP * maxDirections * sizeof(real)));
}

real *GPUMultipleDirectionSolver::Idir(const int direction) {
    assert(direction < maxDirections);
    return Idirs + direction * mv.nP;
}

GPUMultipleDirectionSolver::~GPUMultipleDirectionSolver() {
    CUDA_CHECK(hipFree(Idirs));
}
