#include "gpu.h"
#include "kernels.h"

#include "MeshView.h"

#include <iostream>
#include <cstdlib>
#include <algorithm>
#include <cassert>

#include <hipblas.h>

#define CUDA_CHECK(x) do { hipError_t __err = (x);\
    if (__err != hipSuccess) { std::cerr << __FILE__ << ":" << __LINE__ << " CUDA call `" << #x \
        << "' failed with error `" << hipGetErrorString(__err) << "'" << std::endl; abort(); } \
} while (false)

GPUMeshView::GPUMeshView(int rank, int device, MeshView &mv) {
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, device));
    CUDA_CHECK(hipSetDevice(device));
    std::cout << "Rank " << rank << " using device #" << device << " (" << prop.name << ")" << std::endl;

    nP = mv.pts.size();
    int nT = mv.elems.size();

    CUDA_CHECK(hipMalloc(&pts,    nP * sizeof(point)));
    CUDA_CHECK(hipMalloc(&anyTet, nP * sizeof(int)));
    CUDA_CHECK(hipMalloc(&elems,  nT * sizeof(MeshElement)));

    CUDA_CHECK(hipMemcpy(pts,    mv.pts   .data(), nP * sizeof(point),       hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(anyTet, mv.anyTet.data(), nP * sizeof(int),         hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(elems,  mv.elems .data(), nT * sizeof(MeshElement), hipMemcpyHostToDevice));
}

GPUMeshView::~GPUMeshView() {
    CUDA_CHECK(hipFree(pts));
    CUDA_CHECK(hipFree(anyTet));
    CUDA_CHECK(hipFree(elems));
}

GPUAverageSolution::GPUAverageSolution(const GPUMeshView &gmv) : nP(gmv.nP), U(nP) {
    hipblasStatus_t status = cublasInit();
    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "CUBLAS init failed with status " << status << std::endl;
        abort();
    }
    CUDA_CHECK(hipMalloc(&Udev, nP * sizeof(real)));
    CUDA_CHECK(hipMemset(Udev, 0, nP * sizeof(real)));
}

GPUAverageSolution::~GPUAverageSolution() {
    CUDA_CHECK(hipFree(Udev));
    cublasShutdown();
}

std::vector<double> &GPUAverageSolution::retrieve() {
    std::vector<real> Uhost(nP);
    CUDA_CHECK(hipMemcpy(Uhost.data(), Udev, nP * sizeof(real), hipMemcpyDeviceToHost));
    std::copy(Uhost.begin(), Uhost.end(), U.begin());
    return U;
}

template<>
void GPUAverageSolution::add<float>(float *Idir, const float wei) {
    assert(sizeof(real) == sizeof(float));
    hipblasSaxpy(nP, wei, Idir, 1, (float *)Udev, 1);
}

template<>
void GPUAverageSolution::add<double>(double *Idir, const double wei) {
    assert(sizeof(real) == sizeof(double));
    hipblasDaxpy(nP, wei, Idir, 1, (double *)Udev, 1);
}

GPUMultipleDirectionSolver::GPUMultipleDirectionSolver(
        const int maxDirections, const GPUMeshView &mv, const std::vector<point> &ws
)
    : maxDirections(maxDirections), mv(mv)
{
    CUDA_CHECK(hipMalloc(&Idirs, mv.nP * maxDirections * sizeof(real)));
    CUDA_CHECK(hipMalloc(&inner, mv.nP * maxDirections * sizeof(int)));
    CUDA_CHECK(hipMalloc(&w, ws.size() * sizeof(point)));
    CUDA_CHECK(hipMemcpy(w, ws.data(), ws.size() * sizeof(point), hipMemcpyHostToDevice));
}

real *GPUMultipleDirectionSolver::Idir(const int direction) {
    assert(direction < maxDirections);
    return Idirs + direction * mv.nP;
}

int *GPUMultipleDirectionSolver::innerFlag(const int direction) {
    assert(direction < maxDirections);
    return inner + direction * mv.nP;
}

GPUMultipleDirectionSolver::~GPUMultipleDirectionSolver() {
    CUDA_CHECK(hipFree(Idirs));
    CUDA_CHECK(hipFree(inner));
    CUDA_CHECK(hipFree(w));
}

void GPUMultipleDirectionSolver::setBoundary(
        const int direction, std::vector<real> &Ihostdir, std::vector<int> &isInner)
{
    CUDA_CHECK(hipMemcpy(Idir(direction), Ihostdir.data(), mv.nP * sizeof(real), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(innerFlag(direction), isInner.data(), mv.nP * sizeof(int), hipMemcpyHostToDevice));
}

void GPUMultipleDirectionSolver::traceInterior(const int lo, const int ndir) {
    const int nP = mv.nP;

    dim3 block(256);
    dim3 grid((nP + block.x - 1) / block.x, ndir);

    trace_kernel<<<grid, block>>>(nP, lo, mv, Idirs, inner, w);
}
